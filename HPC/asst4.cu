
#include <bits/stdc++.h>
using namespace std ;
#include <hip/hip_runtime.h>

// dynameic array 
// take n 
// fill array
// make arrr for paralle
// set config
// call s fun
// call p fun

__global__ void addp( int * a , int * b , int * c , int n){
    int tid = blockIdx.x * blockDim.x + threadIdx.x ;
    if(tid < n) c[tid] = a[tid] + b[tid];
}

void add( int * a , int * b , int * c , int n){
    for(int i = 0 ; i < n ; i++)c[i] = a[i] + b[i];
}

void print(int * arr , int n ){
  for(int i = 0 ; i < n ; i++)cout<< arr[i]<<" ";
  cout << endl ;
}

int main(){
  cout<< "Hii1";
  int n ; 
  n = 10 ; 
  int * a = new int[n] ;
  int * b = new int[n] ;
  int * c = new int[n] ;

  // initialize it
  for(int i = 0 ; i < n ; i++)a[i] = rand()%n ;
  for(int i = 0 ; i < n ; i++)b[i] = rand()%n ;
  for(int i = 0 ; i < n ; i++)c[i] = 0 ;

  int * pa , *pb , *pc ;
  hipMalloc( & pa , n * sizeof(int)) ;
  hipMalloc( & pb , n * sizeof(int)) ;
  hipMalloc( & pc , n * sizeof(int)) ;
  hipMemcpy(pa , a , n * sizeof(int) , hipMemcpyHostToDevice ) ;
  hipMemcpy(pb , b , n * sizeof(int) , hipMemcpyHostToDevice ) ; 

  int threadPerBlock = 256 ; 
  int blockPerGrid = (n+ threadPerBlock -1)/threadPerBlock ; 

   print(c , n) ;
  addp<<<blockPerGrid ,threadPerBlock >>>(pa,pb,pc,n) ;
  hipDeviceSynchronize();
   hipMemcpy(c , pc , n * sizeof(int) , hipMemcpyDeviceToHost ) ; 
   print(a,n) ;
   print(b,n) ;
   print(c , n) ;

   add(a,b,c,n) ;
   print(c , n) ;


   delete [] a;
   delete [] b;
   delete [] c;
   hipFree(pa);
   hipFree(pb);
   hipFree(pc);

  return 0 ; 
}